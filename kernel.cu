#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>


__global__ void RGBtoGray (double *gray,double *R,double *G,double *B,int row,int col)
{
	int j=(blockIdx.x)*(blockDim.x)+(threadIdx.x);
	int i=(blockIdx.y)*(blockDim.y)+(threadIdx.y);
	
    if( i<col && j<row )
	   gray[i+j*row]=R[i+j*row]*0.299 + G[i+j*row]*0.587 + B[i+j*row]*0.114;
	//if( i<row && j<col )
	   //gray[i*col+j]=R[i*col+j]*0.299 + G[i*col+j]*0.587 + B[i*col+j]*0.114;
}
