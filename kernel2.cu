#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>


__global__ void refineGradients(double* out,double* in,int row,int col)
{
	//Get individual indices
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	//check bounds and process
	if(i < col-2 && j < row-2)
	{
		//get X gradient
		float GX = in[(i+2)*row + j] + in[(i+2)*row + j+1]*2 +  in[(i+2)*row + j+2]-(in[(i)*row + j] + in[(i)*row + j+1]*2 + in[(i)*row + j+2]);
		
		//get Y gradient
		float GY = in[(i)*row + j+2]+in[(i+1)*row + j+2]*2+in[(i+2)*row + j+2]-(in[(i)*row + j]+in[(i+1)*row + j]*2+in[(i+2)*row + j]);
		
		//calculate final Gradient
		float Gfinal = sqrt((GX*GX)+(GY*GY));
		
		//replace in output matrix
		out[(i)*row + j] = Gfinal;
	}
	return ;
}
